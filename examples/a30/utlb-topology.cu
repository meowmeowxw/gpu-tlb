#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>

#define CHUNK0_SIZE (64L * 1024L * 1024L * 1024L * 1024L + 0x55554000000L)
#define CHUNK1_SIZE (41L * 1024L * 1024L * 1024L * 1024L + 0x0ffc8000000L)
#define STRIDE_SIZE (1L * 1024L * 1024L)

#define BASE_ADDR_SMID0 0x700000000000
#define BASE_ADDR_SMID1 0x702000000000
#define DUMMY_ADDR 0x7F0000000000

#define PAGE0_NUM 8000
#define PAGE1_NUM 30000
#define PAGE_DTLB_NUM 16
#define PAGE_FILL_NUM  6000
#define WAIT_TIME 5000000000L

#define BLK_NUM 100
#define SHARED_MEM (96 * 1024)

uint32_t SMID0 = 0;
uint32_t SMID1 = 1;

uint32_t l1_idx_vec[] = {
  0, 129, 258, 387,
  1, 128, 259, 386,
  2, 131, 256, 385,
  3, 130, 257, 384,
};

__global__ void loop(volatile uint64_t *page0, volatile uint64_t *page1, volatile uint64_t *page_dtlb_smid0, volatile uint64_t *page_dtlb_smid1,
                    volatile uint64_t *page_fill, uint64_t x, uint32_t SMID0, uint32_t SMID1) {
    uint64_t y = x;
    volatile uint64_t *ptr;
    volatile uint64_t *evt;
    uint64_t clk0 = 0;
    uint64_t clk1 = 0;
    uint32_t smid;

    asm("mov.u32 %0, %%smid;" : "=r"(smid));
    if (smid != SMID0 && smid != SMID1)
        return;
    // if (smid != SMID0)
    //   return;

    if (smid == SMID0) {
        for (ptr = (uint64_t *)page_fill[0]; ptr != page_fill; ptr = (uint64_t *)ptr[0]) {
            ++ptr[2];
        }
        while (y == x) {
            // printf("l2: accessing %p, page0[0]: %p\n", page0, page0[0]);
            // Fill L2-uTLB
            for (ptr = (uint64_t *)page0[0]; ptr != page0; ptr = (uint64_t *)ptr[0]) {
                ++ptr[2];
                // Fill L1-dTLB
                for (evt = (uint64_t *)page_dtlb_smid0[0]; evt != page_dtlb_smid0; evt = (uint64_t *)evt[0]) {
                    ++evt[2];
                }
            }

            printf("timer routine\n");
            // Wait for PTE modification
            clk0 = clock64();
            clk1 = 0;
            while (clk1 < WAIT_TIME)
                clk1 = clock64() - clk0;

            // Reaccess page0[0], if changed it was evicted from TLB hierarchy
            y = ptr[1];
            printf("[0] y: %lx\n", y);
        }
    } else if (smid == SMID1) {
        // ptr = (uint64_t *)page1[0];
        // Fill L2/L3-uTLB
        while (y == x) {
            ptr = (uint64_t *)page1[0];
            do {
                ++ptr[2];
                ptr = (uint64_t *)ptr[0];
                // Fill L1-dTLB
                for (evt = (uint64_t *)page_dtlb_smid1[0]; evt != page_dtlb_smid1; evt = (uint64_t *)evt[0]) {
                    ++evt[2];
                }
            } while (ptr != page1);
            y = page1[1];
        }
        printf("[1] y: %lx\n", y);
    }
    page0[1] = 0;
    page1[1] = 0;
}

int main(int argc, char *argv[]) {
    uint8_t *chunk0 = NULL;
    uint8_t *chunk1 = NULL;
    uint8_t *base = NULL;
    uint64_t **list_smid0 =      (uint64_t **)_malloc(sizeof(uint64_t *) * PAGE0_NUM);
    uint64_t **list_smid1 =      (uint64_t **)_malloc(sizeof(uint64_t *) * PAGE1_NUM);
    uint64_t **list_dtlb_smid0 = (uint64_t **)_malloc(sizeof(uint64_t *) * PAGE_DTLB_NUM);
    uint64_t **list_dtlb_smid1 = (uint64_t **)_malloc(sizeof(uint64_t *) * PAGE_DTLB_NUM);
    uint64_t **list_fill =       (uint64_t **)_malloc(sizeof(uint64_t *) * (PAGE_FILL_NUM + 1));
    struct __eviction_set es_smid0;
    struct __eviction_set es_smid1;

    int aim = -1;
    uint64_t *dummy = NULL;

    parse_eviction_set("./out/eviction_set_smid0.txt", &es_smid0);
    parse_eviction_set("./out/eviction_set_smid1.txt", &es_smid1);

    hipDeviceReset();
    hipFuncSetAttribute(reinterpret_cast<const void*>(loop), hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_MEM);

    // hoard a large address space
    hipMallocManaged(&chunk0, CHUNK0_SIZE);

    hipMallocManaged(&chunk1, CHUNK1_SIZE);

    SMID0 = atoi(argv[1]);
    SMID1 = atoi(argv[2]);

    aim = es_smid0.indexes[0];

    base = (uint8_t *)BASE_ADDR_SMID0;
    for (int i = 0; i < PAGE0_NUM; ++i)
        list_smid0[i] = (uint64_t *)(base + i * STRIDE_SIZE);
    base = (uint8_t *)BASE_ADDR_SMID1;
    for (int i = 0; i < PAGE1_NUM; ++i)
        list_smid1[i] = (uint64_t *)(base + i * STRIDE_SIZE);
    for (int i = 0; i < PAGE_DTLB_NUM; ++i) {
        list_dtlb_smid0[i] = list_smid0[l1_idx_vec[i]];
        list_dtlb_smid1[i] = list_smid1[l1_idx_vec[i]];
    }
    for (int i = 1; i < PAGE_FILL_NUM + 1; ++i) {
        list_fill[i] = (uint64_t *)((uint8_t *)(DUMMY_ADDR) + i * STRIDE_SIZE);
    }
    dummy = (uint64_t *)DUMMY_ADDR;

    put<<<1, 1>>>(dummy, 0, 0);

    for (int i = 0; i < es_smid0.count; ++i) {
        int m = es_smid0.indexes[i];
        int n = es_smid0.indexes[(i + 1) % es_smid0.count];
        put<<<1, 1>>>(list_smid0[m], (uint64_t)list_smid0[n], 0xdeadbeef);
        printf("[smid0] m: 0x%lx, n: 0x%lx\n", (uint64_t)list_smid0[m], (uint64_t)list_smid0[n]);
    }
    for (int i = 0; i < es_smid1.count; ++i) {
        int m = es_smid1.indexes[i];
        int n = es_smid1.indexes[(i + 1) % es_smid1.count];
        put<<<1, 1>>>(list_smid1[m], (uint64_t)list_smid1[n], 0xdeadbeef);
        printf("[smid1] m: 0x%lx, n: 0x%lx\n", (uint64_t)list_smid1[m], (uint64_t)list_smid1[n]);
    }
    for (int i = 0; i < PAGE_DTLB_NUM; ++i) {
        put<<<1, 1>>>(list_dtlb_smid0[i], (uint64_t)list_dtlb_smid0[(i + 1) % PAGE_DTLB_NUM], 1);
        put<<<1, 1>>>(list_dtlb_smid1[i], (uint64_t)list_dtlb_smid1[(i + 1) % PAGE_DTLB_NUM], 2);
    }
    for (int i = 1; i < PAGE_FILL_NUM + 1; ++i) {
        int j = (i + 1) % PAGE_FILL_NUM == 0 ? 1 : i + 1;
        put<<<1, 1>>>(list_fill[i], (uint64_t)list_fill[j], 3);
    }
    hipDeviceSynchronize();

    printf("Done hoarding, aim: 0x%x, page: 0x%lx, smid: 0x%x\n", aim, (uint64_t)list_smid0[aim], SMID1);
    loop<<<BLK_NUM, 1, SHARED_MEM>>>(list_smid0[aim], list_smid1[es_smid1.indexes[0]], list_dtlb_smid0[0],
                                     list_dtlb_smid1[0], list_fill[1], 0xdeadbeef, SMID0, SMID1);
    // loop<<<BLK_NUM, 1, SHARED_MEM>>>(list_smid0[aim], list_smid1[atoi(argv[2])], list_dtlb[0], 0xdeadbeef);
    hipDeviceSynchronize();

    hipFree(chunk0);
    hipFree(chunk1);

    free(es_smid0.indexes);
    free(es_smid1.indexes);
}
